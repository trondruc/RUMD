#include "hip/hip_runtime.h"
/*
    Copyright (C) 2010  Thomas Schrøder

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    LICENSE.txt file for license details.
*/

#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/gather.h>

#include "rumd/RUMD_Error.h"
#include "rumd/ParticleData.h"
#include "rumd/Device.h"

//////////////////////////////////////////////////
// CONSTRUCTION/DESTRUCTION 
//////////////////////////////////////////////////

ParticleData::ParticleData() :   numParticles(0),
				 numVirtualParticles(0),
				 allocatedNumParticles(0),
				 numBlocks(0),
				 numberOfType(),
				 massOfType()
{

}

ParticleData::~ParticleData(){
  FreeParticles();
}

// This copies device data only!
ParticleData& ParticleData::operator=(const ParticleData& P){
  if(this != &P){ 
    if(numParticles != P.GetNumberOfParticles())
      SetNumberOfParticles(P.GetNumberOfParticles(), P.GetNumberOfVirtualParticles()/P.GetNumberOfBlocks());

    hipMemcpy( d_r, P.d_r, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_v, P.d_v, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_f, P.d_f, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_w, P.d_w, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_im, P.d_im, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_sts, P.d_sts, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_misc, P.d_misc, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_unsorted_index, P.d_unsorted_index, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice);
  };
  return *this;   
}

// Copy only numParticles items in each array in case the other object
// has a different pb and hence number of virtual particles
void ParticleData::CopyHostData(const ParticleData& P) {
  if(numParticles != P.GetNumberOfParticles())
    SetNumberOfParticles(P.GetNumberOfParticles(), P.GetNumberOfVirtualParticles()/P.GetNumberOfBlocks());
  
  hipMemcpy( h_r, P.h_r, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_v, P.h_v, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_f, P.h_f, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_w, P.h_w, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_im, P.h_im, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_sts, P.h_sts, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_misc, P.h_misc, numParticles * sizeof(float4), hipMemcpyHostToHost );
  hipMemcpy( h_Type, P.h_Type, numParticles * sizeof(unsigned int), hipMemcpyHostToHost );

  SetNumberOfTypes(P.GetNumberOfTypes());

  for(unsigned int type = 0; type < GetNumberOfTypes(); type++)  {
    SetMass(type, P.GetMass(type));
    SetNumberThisType(type, P.GetNumberThisType(type));
  }
}

//////////////////////////////////////////////////
// MEMORY ALLOCATION/DEALLOCATION 
//////////////////////////////////////////////////

void ParticleData::SetNumberOfParticles(unsigned int set_numParticles, unsigned int pb) {
 
  numBlocks = (set_numParticles + pb - 1) / pb;    
  numVirtualParticles = numBlocks * pb;

  if(numBlocks > Device::GetDevice().GetMaximumGridDimensionX() )
    throw RUMD_Error("ParticleData",__func__, "Too many blocks for this device");
  bool copy_data = false;
  // we reallocate and copy data if the number of particles is unchanged
  // but the number of virtual particles is changed
  if(numParticles > 0 && set_numParticles == numParticles 
     &&  allocatedNumParticles != numVirtualParticles)
    copy_data = true;

  numParticles = set_numParticles;


  if( !( (numParticles <= pb*numBlocks) && (numVirtualParticles >= numParticles) && (numVirtualParticles <  (numParticles + pb)) ) )
    throw(RUMD_Error("ParticleData","SetNumberOfParticles","ParticleData could not validate internal variables. Report this error to RUMD developers"));
  
  // Allocate memory.
  AllocateParticles(numVirtualParticles, copy_data);
}

void ParticleData::AllocateParticles(unsigned int nvp, bool copy_data){
  if( allocatedNumParticles == nvp )
    return;
  
  // for saving the data (ie if only reallocating because pb has changed)
  // we need some extra pointers
  float4* h_r_new = 0; 
  float4* h_v_new = 0; 
  float4* h_f_new = 0;
  float4* h_w_new = 0;
  float4* h_im_new = 0;
  float4* h_sts_new = 0;
  float4* h_misc_new = 0;
  unsigned int* h_Type_new = 0;

  float4* d_r_new = 0; 
  float4* d_v_new = 0; 
  float4* d_f_new = 0;
  float4* d_w_new = 0;
  float4* d_im_new = 0;
  float4* d_sts_new = 0;
  float4* d_misc_new = 0; 
  float4* d_temp_new = 0; 
  unsigned int* d_unsorted_index_new = 0;
  unsigned int* d_temp_uint_new = 0; 

  // if not saving we free the memory first, to cut down on memory use
  if(!copy_data)
    FreeParticles();

  // Page-locked CPU Allocation
  if( hipHostMalloc( (void**) &h_r_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_r") );
  
  if( hipHostMalloc( (void**) &h_v_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_v") );
  
  if( hipHostMalloc( (void**) &h_f_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_f") );
  
  if( hipHostMalloc( (void**) &h_w_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_w") );
  
  if( hipHostMalloc( (void**) &h_im_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_im") );
  
  if( hipHostMalloc( (void**) &h_sts_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_sts") );
  
  if( hipHostMalloc( (void**) &h_misc_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_misc") );
  
  if( hipHostMalloc( (void**) &h_Type_new, nvp * sizeof(unsigned int) ) == hipErrorOutOfMemory ) 
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on h_Type") );
  
  // GPU Allocation
  if( hipMalloc( (void**) &d_r_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_r") );

  if( hipMalloc( (void**) &d_v_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_v") );
  
  if( hipMalloc( (void**) &d_f_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_f") );
  
  if( hipMalloc( (void**) &d_w_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_w") );
  
  if( hipMalloc( (void**) &d_im_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_im") );
  
  if( hipMalloc( (void**) &d_sts_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_sts") );
  
  if( hipMalloc( (void**) &d_misc_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_misc") );

  if( hipMalloc( (void**) &d_temp_new, nvp * sizeof(float4) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_temp") );

  if( hipMalloc( (void**) &d_unsorted_index_new, nvp * sizeof(unsigned int) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_unsorted_index") );

  if( hipMalloc( (void**) &d_temp_uint_new, nvp * sizeof(unsigned int) ) == hipErrorOutOfMemory )
    throw( RUMD_Error("ParticleData","AllocateParticles","Malloc failed on d_temp_uint") );

  // Initialize all CPU memory to zero from the start.
  memset( h_r_new,      0, nvp * sizeof(float4) );
  memset( h_v_new,      0, nvp * sizeof(float4) );
  memset( h_f_new,      0, nvp * sizeof(float4) );
  memset( h_w_new,      0, nvp * sizeof(float4) );
  memset( h_im_new,     0, nvp * sizeof(float4) );
  memset( h_sts_new,    0, nvp * sizeof(float4) );
  memset( h_misc_new,   0, nvp * sizeof(float4) );
  memset( h_Type_new,   0, nvp * sizeof(unsigned int) );

  // Initialize all GPU memory to zero from the start (needed+safety).
  hipMemset( d_r_new,              0, nvp * sizeof(float4) );
  hipMemset( d_v_new,              0, nvp * sizeof(float4) );
  hipMemset( d_f_new,              0, nvp * sizeof(float4) );
  hipMemset( d_w_new,              0, nvp * sizeof(float4) );
  hipMemset( d_im_new,             0, nvp * sizeof(float4) );
  hipMemset( d_sts_new,            0, nvp * sizeof(float4) );
  hipMemset( d_misc_new,           0, nvp * sizeof(float4) );
  hipMemset( d_temp_new,           0, nvp * sizeof(float4) );
  hipMemset( d_unsorted_index_new, 0, nvp * sizeof(unsigned int) );
  hipMemset( d_temp_uint_new,     0, nvp * sizeof(unsigned int) );

  if( hipDeviceSynchronize() != hipSuccess )
    throw( RUMD_Error("ParticleData","AllocateParticles","Initialization failed on GPU") );

  if(copy_data) {
    // copy from old arrays
    hipMemcpy( h_r_new, h_r, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_v_new, h_v, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_f_new, h_f, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_w_new, h_w, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_im_new, h_im, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_sts_new, h_sts, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_misc_new, h_misc, numParticles * sizeof(float4), hipMemcpyHostToHost );
    hipMemcpy( h_Type_new, h_Type, numParticles * sizeof(unsigned int), hipMemcpyHostToHost );
    

    hipMemcpy( d_r_new, d_r, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_v_new, d_v, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_f_new, d_f, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_w_new, d_w, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_im_new, d_im, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_sts_new, d_sts, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_misc_new, d_misc, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_temp_new, d_temp, numParticles * sizeof(float4), hipMemcpyDeviceToDevice );
    hipMemcpy( d_unsorted_index_new, d_unsorted_index, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice );
    hipMemcpy( d_temp_uint_new, d_temp_uint, numParticles * sizeof(unsigned int), hipMemcpyDeviceToDevice );
 
    // Free old arrays
    FreeParticles();
  }

  // reassign the main pointers to the newly allocated arrays
  h_r = h_r_new;
  h_v = h_v_new;
  h_f = h_f_new;
  h_w = h_w_new;
  h_im = h_im_new;
  h_sts = h_sts_new;
  h_misc = h_misc_new;
  h_Type = h_Type_new;

  d_r = d_r_new;
  d_v = d_v_new;
  d_f = d_f_new;
  d_w = d_w_new;
  d_im = d_im_new;
  d_sts = d_sts_new;
  d_misc = d_misc_new;
  d_temp = d_temp_new;
  d_unsorted_index = d_unsorted_index_new;
  d_temp_uint = d_temp_uint_new;

  allocatedNumParticles = nvp;
}

void ParticleData::FreeParticles() {
  if(allocatedNumParticles == 0)
    return;
  
  hipHostFree(h_r);
  hipHostFree(h_v);
  hipHostFree(h_f);
  hipHostFree(h_w);
  hipHostFree(h_im);
  hipHostFree(h_sts);
  hipHostFree(h_misc);
  hipHostFree(h_Type);
  hipFree(d_r);
  hipFree(d_v);
  hipFree(d_f);
  hipFree(d_w);
  hipFree(d_im);
  hipFree(d_sts);  
  hipFree(d_misc);
  hipFree(d_temp);
  hipFree(d_unsorted_index);
  hipFree(d_temp_uint);
}

void ParticleData::SetAllMasses( double* mass_array, int length ) {
  if(length != (int) numberOfType.size())
    throw RUMD_Error("ParticleData", __func__, "Wrong length array passed");
  for(int idx=0;idx< length;idx++)
    massOfType[idx] = mass_array[idx];

  UpdateParticleMasses();
}
  

void ParticleData::UpdateParticleMasses() {
  if(numParticles) {
    for(unsigned idx=0; idx<numParticles;idx++)
      h_v[idx].w = 1.f/massOfType[h_Type[idx]];

    CopyVelToDevice();
  }
}


void ParticleData::SetForcesToZero() const {
  // Also potential energy, virial, stresses.
  // To be used as initialization by some potentials.

  // Note that const-ness is not "deep"; const here means the pointers cannot
  // be changed, while the data in the arrays can be
  hipMemset( d_f, 0, numVirtualParticles * sizeof(float4) );
  hipMemset( d_w, 0, numVirtualParticles * sizeof(float4) );
  hipMemset( d_sts, 0, numVirtualParticles * sizeof(float4) );

}


//////////////////////////////////////////////////
// Copy to the device.
//////////////////////////////////////////////////

void ParticleData::CopyPosToDevice(bool reset_sorting) const{

  if(reset_sorting) {
    hipMemcpy( d_r, h_r, numVirtualParticles * sizeof(float4), hipMemcpyHostToDevice );
    
    // reset unsorted_index - happens in default case, for example when reading
    // in a new configuration from a file. It must be assumed that fresh
    // velocities are also being read in from the file.
    thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
    thrust::sequence(thrust_d_unsorted_index, thrust_d_unsorted_index + numParticles);
  }
  else
    {
      // distribute according to existing sorting, for example when using
      // SetPositions from python
      hipMemcpy( d_temp, h_r, numVirtualParticles * sizeof(float4), hipMemcpyHostToDevice );
      thrust::device_ptr<float4> thrust_d_r(d_r);
      thrust::device_ptr<float4> thrust_d_temp(d_temp);
      thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
      thrust::gather(thrust_d_unsorted_index, thrust_d_unsorted_index+numParticles, thrust_d_temp, thrust_d_r);      
    }    
}

void ParticleData::CopyVelToDevice() const{
  hipMemcpy( d_v, h_v, numVirtualParticles * sizeof(float4), hipMemcpyHostToDevice );
}

void ParticleData::CopyForToDevice() const{
  hipMemcpy( d_f, h_f, numVirtualParticles * sizeof(float4), hipMemcpyHostToDevice );
}

void ParticleData::CopyImagesToDevice() const{
  hipMemcpy( d_im, h_im, numVirtualParticles * sizeof(float4), hipMemcpyHostToDevice );
}

void ParticleData::CopyConfToDevice() const{
  CopyPosToDevice();
  CopyVelToDevice();
  CopyForToDevice();
  CopyImagesToDevice();
  
  // Check if the copy generated errors.
  if( hipDeviceSynchronize() != hipSuccess ) 
    throw( RUMD_Error("ParticleData", "CopyConfToDevice", "hipMemcpy failed: simulation state => device") );
}

//////////////////////////////////////////////////
// Copy from the device.
//////////////////////////////////////////////////

void ParticleData::CopyPosFromDevice(bool sync) const{
  //hipMemcpy( h_r, d_r, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_r(d_r);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_r, thrust_d_r + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_r, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_r, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
}

void ParticleData::CopyPosImagesDevice(float4* d_r_dest, float4* d_im_dest) const {
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_r(d_r);
  thrust::device_ptr<float4> thrust_d_r_dest(d_r_dest);
  thrust::scatter(thrust_d_r, thrust_d_r + numParticles, thrust_d_unsorted_index, thrust_d_r_dest);

  thrust::device_ptr<float4> thrust_d_im(d_im);
  thrust::device_ptr<float4> thrust_d_im_dest(d_im_dest);
  thrust::scatter(thrust_d_im, thrust_d_im + numParticles, thrust_d_unsorted_index, thrust_d_im_dest);
}

void ParticleData::CopyVelFromDevice(bool sync) const{
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_v(d_v);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_v, thrust_d_v + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_v, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_v, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
}

void ParticleData::CopyForFromDevice(bool sync) const{
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_f(d_f);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_f, thrust_d_f + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_f, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_f, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );

}

void ParticleData::CopyVirFromDevice(bool sync) const{
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_w(d_w);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_w, thrust_d_w + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_w, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_w, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );

}

void ParticleData::CopyImagesFromDevice(bool sync) const{
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_im(d_im);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_im, thrust_d_im + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_im, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_im, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );

}

void ParticleData::CopyStressFromDevice(bool sync) const{
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_sts(d_sts);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_sts, thrust_d_sts + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_sts, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_sts, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
}

void ParticleData::CopyMiscFromDevice(bool sync) const{ 
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<float4> thrust_d_misc(d_misc);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::scatter(thrust_d_misc, thrust_d_misc + numParticles, thrust_d_unsorted_index, thrust_d_temp);
  if(sync)
    hipMemcpy( h_misc, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
  else
    hipMemcpyAsync( h_misc, d_temp, numVirtualParticles * sizeof(float4), hipMemcpyDeviceToHost );
}

void ParticleData::CopyConfFromDevice(bool sync) const{
  CopyPosFromDevice(sync);
  CopyVelFromDevice(sync);
  CopyForFromDevice(sync);
  CopyVirFromDevice(sync);
  CopyImagesFromDevice(sync);
  CopyStressFromDevice(sync);
  CopyMiscFromDevice(sync);

  // Check if the copy generated errors; make sure asynchronous copies finish
  if( hipDeviceSynchronize() != hipSuccess ) 
    throw( RUMD_Error("ParticleData","CopyConfFromDevice","hipMemcpy failed: simulation state => host") );
}

//////////////////////////////////////////////
// Sorting of particles on device.
//////////////////////////////////////////////

void ParticleData::UpdateAfterSorting(thrust::device_vector<unsigned int>&  thrust_old_index) {

  // wrap raw pointers with thrust device_ptrs 
  thrust::device_ptr<float4> thrust_d_r(d_r);
  thrust::device_ptr<float4> thrust_d_im(d_im);
  thrust::device_ptr<float4> thrust_d_v(d_v);
  thrust::device_ptr<float4> thrust_d_f(d_f);
  thrust::device_ptr<float4> thrust_d_w(d_w);
  thrust::device_ptr<float4> thrust_d_misc(d_misc);
  thrust::device_ptr<float4> thrust_d_temp(d_temp);
  thrust::device_ptr<unsigned int> thrust_d_unsorted_index(d_unsorted_index);
  thrust::device_ptr<unsigned int> thrust_d_temp_uint(d_temp_uint);

  // Move all particle data to be consistent with new sorted order (Gather + Swap instead?)
  thrust::copy(thrust_d_r, thrust_d_r + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_r);

  thrust::copy(thrust_d_im, thrust_d_im + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_im);
  
  thrust::copy(thrust_d_v, thrust_d_v + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_v);

  thrust::copy(thrust_d_f, thrust_d_f + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_f);

  thrust::copy(thrust_d_w, thrust_d_w + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_w);

  thrust::copy(thrust_d_misc, thrust_d_misc + numParticles, thrust_d_temp);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp, thrust_d_misc);
  
  thrust::copy(thrust_d_unsorted_index, thrust_d_unsorted_index + numParticles, thrust_d_temp_uint);
  thrust::gather(thrust_old_index.begin(), thrust_old_index.end(), thrust_d_temp_uint, thrust_d_unsorted_index);

}



//////////////////////////////////////////////////
// Work on device data
//////////////////////////////////////////////////

void ParticleData::IsotropicScalePositions( float Rscal ){
  dim3 numBlocks( (numParticles+32-1)/32 );
  IsotropicScalePositionsKernel<<<numBlocks, 32>>>( d_r, numParticles, Rscal );
}


void ParticleData::AnisotropicScalePositions( float Rscal, unsigned dir ){
  dim3 numBlocks( (numParticles+32-1)/32 );
  AnisotropicScalePositionsKernel<<<numBlocks, 32>>>( d_r, numParticles, Rscal, dir );
}

void ParticleData::AffinelyShearPositions(float shear_strain) {
  dim3 numBlocks( (numParticles+32-1)/32 );
  AffinelyShearPositionsKernel<<<numBlocks, 32>>>(d_r, numParticles, shear_strain);
}

void ParticleData::ScaleVelocities(float factor) {
  dim3 numBlocks( (numParticles+32-1)/32 );
  ScaleVelocitiesKernel<<<numBlocks, 32>>>( d_v, numParticles, factor );
}						  


void ParticleData::ApplyLeesEdwardsWrapToImages(float wrap) 
{
  dim3 numBlocks( (numParticles+32-1)/32 );
  ApplyLeesEdwardsWrapToImagesKernel<<<numBlocks, 32>>>( d_im,
							 numParticles,
							 wrap );
}

// Kernels

__global__ void IsotropicScalePositionsKernel( float4 *r, unsigned numParticles,
					       float Rscal ){
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if ( i < numParticles ) {
    float4 my_r = r[i];
    my_r.x *= Rscal;
    my_r.y *= Rscal;
    my_r.z *= Rscal;
    r[i] = my_r;
  }
}


__global__ void AnisotropicScalePositionsKernel( float4 *r, unsigned numParticles, 
						 float Rscal, unsigned dir ){
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if ( i < numParticles ) {
    float4 my_r = r[i];
    if ( dir == 0 )
      my_r.x *= Rscal;
    else if ( dir == 1 )
      my_r.y *= Rscal;
    else if ( dir == 2 )
      my_r.z *= Rscal;
    r[i] = my_r;
  }
}

__global__ void AffinelyShearPositionsKernel(float4 *r, unsigned numParticles, float shear_strain) {
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  if ( i < numParticles ) {
    float4 my_r = r[i];
    my_r.x += shear_strain*my_r.y;  
    r[i] = my_r;
  }
}

__global__ void ScaleVelocitiesKernel( float4 *v, unsigned numParticles, float factor ){
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if ( i < numParticles ) {
    float4 my_v = v[i];
    my_v.x *= factor;
    my_v.y *= factor;
    my_v.z *= factor;
    v[i] = my_v;
  }
}



__global__ void ApplyLeesEdwardsWrapToImagesKernel( float4 *image, unsigned numParticles, float wrap )
{
  unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if ( i < numParticles ) {
    float4 my_image = image[i];
    my_image.x += wrap * my_image.y;

    image[i] = my_image;
  }
}
