/*
    Copyright (C) 2010  Thomas Schrøder

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    LICENSE.txt file for license details.
*/
#include "rumd/Device.h"

#include <iostream>
#include <cstdlib>
#include <vector>
#include <sys/utsname.h>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include "rumd/RUMD_Error.h"



Device::Device() {
  hipEventCreate( &event1 );
  hipEventCreate( &event2 );
  hipEventRecord( event1, 0);
  hipEventSynchronize( event1 );
}

Device::~Device() {
// We used to do a hipDeviceReset() at this point. Destruction of static objects
// happens too late, so the CUDA device may already be released

  hipEventDestroy(event1);
  hipEventDestroy(event2);


}

std::string Device::Report()
{
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");
  hipDeviceProp_t deviceProp;
  hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
  if (err != hipSuccess)
    throw RUMD_Error("Device","Report",std::string("[hipGetDeviceProperties] ")+hipGetErrorString(err));

  struct utsname buf;
  uname(&buf); 
  std::ostringstream device_report;

  device_report
    << " node=" << buf.nodename
    << " device=" << dev
    << " name=\"" << deviceProp.name << "\"";

  return device_report.str();
}

const std::string Device::GetDeviceName() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");
  hipDeviceProp_t deviceProp;
  hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
  if (err != hipSuccess)
    throw RUMD_Error("Device", __func__, std::string("[hipGetDeviceProperties] ")+hipGetErrorString(err));
  std::ostringstream outputStr;
  outputStr << deviceProp.name;
  return outputStr.str();
}


const std::string Device::GetDeviceReport() {  
  return GetDevice().Report();
}

Device& Device::GetDevice() {
  static Device instance;
  return instance;
}

unsigned Device::GetMaxThreadsPerBlock() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  return deviceProp.maxThreadsPerBlock;
}

unsigned Device::GetMaximumGridDimensionX() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");
  int result;
  hipDeviceGetAttribute(&result , hipDeviceAttributeMaxGridDimX, dev);
  return (unsigned) result;
}

unsigned Device::GetComputeCapability() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");
  hipDeviceProp_t deviceProp;
  hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
  if (err != hipSuccess)
    throw RUMD_Error("Device", __func__, std::string("[hipGetDeviceProperties] ")+hipGetErrorString(err));
  
  unsigned cc = 100 * deviceProp.major + 10 * deviceProp.minor;
  return cc;
}

size_t Device::GetSharedMemPerBlock() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");
  hipDeviceProp_t deviceProp;
  hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
  if (err != hipSuccess)
    throw RUMD_Error("Device", __func__, std::string("[hipGetDeviceProperties] ")+hipGetErrorString(err));
  return deviceProp.sharedMemPerBlock;
}


size_t Device::GetDeviceMemory() {
  int dev;
  if (hipSuccess != hipGetDevice(&dev))
    throw RUMD_Error("Device", __func__, "Cannot get cuda device");

  hipDeviceProp_t deviceProp;
  hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
  if (err != hipSuccess)
    throw RUMD_Error("Device", __func__, std::string("[hipGetDeviceProperties] ")+hipGetErrorString(err));
  return deviceProp.totalGlobalMem;
}



void Device::Init()
{
  int deviceCount;
  hipError_t err = hipGetDeviceCount(&deviceCount);
  if (err != hipSuccess)
    throw RUMD_Error("Device","Init",std::string("[hipGetDeviceCount]") + hipGetErrorString(err));

  if (deviceCount == 0)
    throw RUMD_Error("Device","Init","error: no devices supporting CUDA.");

  if (deviceCount == 1) {
    // just one device or an emulated device present, no choice
    int dev=0;
    hipDeviceProp_t deviceProp;
    hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
    if (err != hipSuccess)
      throw RUMD_Error("Device","Init",std::string("[hipGetDeviceProperties]") + hipGetErrorString(err));
    if (deviceProp.major < 1) 
      throw RUMD_Error("Device","Init","error: device does not support CUDA.");
    if ((deviceProp.major == 1) )// && (deviceProp.minor < 3))
      throw RUMD_Error("Device","Init","error: device compute capability >= 2.0 needed");
    hipSetDevice(dev);
  }
  else {
    // several devices present, so make list of usable devices
    // and have one choosen among the currently available ones
    std::vector<int> usable_devices;
    for (int dev=0; dev<deviceCount; dev++) { 
      hipDeviceProp_t deviceProp;
      hipError_t err = hipGetDeviceProperties(&deviceProp, dev);
      if (err != hipSuccess)
	throw RUMD_Error("Device","Init",std::string("[hipGetDeviceProperties] ") + hipGetErrorString(err));
      if (((deviceProp.major > 1) 
	   //|| ((deviceProp.major == 1) && (deviceProp.minor >= 3))
	   ) && 
	(deviceProp.multiProcessorCount >= 2) &&
	 (deviceProp.computeMode != hipComputeModeProhibited)) {
        usable_devices.push_back(dev); 
      }
    }
    if (usable_devices.size() == 0)
      throw RUMD_Error("Device","Init","error: no usable devices supporting CUDA.");
    hipError_t err = hipSetValidDevices(&usable_devices[0], usable_devices.size());
    if (err != hipSuccess )
      throw RUMD_Error("Device","Init",std::string("[hipSetValidDevices] ") + hipGetErrorString(err));
    // trigger device initialization by a non-device management function call
    hipError_t err2 = hipDeviceSynchronize();
    if (err2 != hipSuccess )
      throw RUMD_Error("Device","Init",std::string("[hipDeviceSynchronize] ") + hipGetErrorString(err));

  }

  // test for incorrect compute capability (thrust is useful for this
  // since it immediately raises an exception; an ordinary kernel will simply
  // not run but give no exception)
  thrust::device_vector<float> thrust_temp(1000);
  thrust::sequence(thrust_temp.begin(), thrust_temp.end());
  
  std::cout << "CUDA:" << Report() << std::endl;
}

void Device::Synchronize() {
  hipDeviceSynchronize();
}

void Device::CheckErrors() {
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
    if(err != hipSuccess)
  	throw( RUMD_Error("Device","CheckErrors", hipGetErrorString(err) ) );
}

float Device::Time() {

  float elapsed_time = 0.f;
  hipEventRecord( event2, 0 );
  hipEventSynchronize( event2 );
  hipEventElapsedTime( &elapsed_time, event1, event2);
  return elapsed_time/1000.; // return time in seconds
}
